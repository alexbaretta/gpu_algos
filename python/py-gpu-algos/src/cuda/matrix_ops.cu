#include "hip/hip_runtime.h"
/*
 * CUDA Matrix Operations Bindings for py-gpu-algos
 *
 * This module provides Python bindings for matrix operations using pybind11.
 * It instantiates the matrix_product_naive kernel for all supported numeric types.
 */

// #ifdef WITH_CUDA

#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>
#include <string>
#include <stdexcept>

// Include CUDA kernel headers
#include "cuda/kernels/matrix_product/matrix_product_naive.cuh"
#include "cuda/kernels/matrix_product/matrix_product_tiled.cuh"
#include "cuda/kernels/matrix_product/matrix_product_warp.cuh"
#include "cuda/kernels/matrix_product/matrix_product_cublas.cuh"
#include "cuda/kernels/matrix_product/matrix_product_cutlass.cuh"
#include "cuda/kernels/matrix_product/matrix_product_tensor.cuh"
#include "cuda/kernels/matrix_transpose/matrix_transpose_striped.cuh"
#include "cuda/kernels/matrix_transpose/matrix_transpose_tiled.cuh"
#include "cuda/cuda_utils.cuh"
#include "cuda/check_errors.cuh"

namespace py = pybind11;

// Helper function to launch matrix_product_naive kernels
template<typename T>
py::array_t<T> matrix_product_naive_cuda_impl(
    const py::array_t<T>& a,
    const py::array_t<T>& b
) {
    // Validate input arrays
    auto a_buf = a.request();
    auto b_buf = b.request();

    if (a_buf.ndim != 2 || b_buf.ndim != 2) {
        throw std::invalid_argument("Input arrays must be 2-dimensional");
    }

    if (!a.flags() & py::array::c_style) {
        throw std::invalid_argument("Array a must be C-contiguous");
    }

    if (!b.flags() & py::array::c_style) {
        throw std::invalid_argument("Array b must be C-contiguous");
    }

    // Get matrix dimensions
    long m = a_buf.shape[0];  // rows of A
    long n = a_buf.shape[1];  // cols of A, rows of B
    long k = b_buf.shape[1];  // cols of B

    if (b_buf.shape[0] != n) {
        throw std::invalid_argument("Matrix dimensions incompatible for multiplication");
    }

    // Create output array
    auto result = py::array_t<T>({m, k});
    auto result_buf = result.request();

    // Get data pointers
    const T* a_ptr = static_cast<const T*>(a_buf.ptr);
    const T* b_ptr = static_cast<const T*>(b_buf.ptr);
    T* result_ptr = static_cast<T*>(result_buf.ptr);

    // Create kernel specification
    Matrix_product_naive_spec spec(
        "float",  // Type string (will be overridden by template parameter)
        m, n, k,
        16, 16    // Default block dimensions
    );

    // Create kernel instance
    Matrix_product_naive_kernel<T> kernel(spec);

    // Allocate GPU memory
    T* d_a = nullptr;
    T* d_b = nullptr;
    T* d_result = nullptr;

    size_t size_a = m * n * sizeof(T);
    size_t size_b = n * k * sizeof(T);
    size_t size_result = m * k * sizeof(T);

    cuda_check_error(hipMalloc(&d_a, size_a), "hipMalloc for matrix A");
    cuda_check_error(hipMalloc(&d_b, size_b), "hipMalloc for matrix B");
    cuda_check_error(hipMalloc(&d_result, size_result), "hipMalloc for result matrix");

    try {
        // Copy data to device
        cuda_check_error(hipMemcpy(d_a, a_ptr, size_a, hipMemcpyHostToDevice), "hipMemcpy A to device");
        cuda_check_error(hipMemcpy(d_b, b_ptr, size_b, hipMemcpyHostToDevice), "hipMemcpy B to device");

        // Create CUDA stream
        hipStream_t stream;
        cuda_check_error(hipStreamCreate(&stream), "hipStreamCreate");

        // Run kernel
        kernel.run_device_kernel(d_a, d_b, d_result, nullptr, stream);

        // Wait for completion
        cuda_check_error(hipStreamSynchronize(stream), "hipStreamSynchronize");

        // Copy result back to host
        cuda_check_error(hipMemcpy(result_ptr, d_result, size_result, hipMemcpyDeviceToHost), "hipMemcpy result to host");

        // Cleanup stream
        cuda_check_error(hipStreamDestroy(stream), "hipStreamDestroy");

    } catch (...) {
        // Cleanup on error
        if (d_a) hipFree(d_a);
        if (d_b) hipFree(d_b);
        if (d_result) hipFree(d_result);
        throw;
    }

    // Cleanup GPU memory
    cuda_check_error(hipFree(d_a), "hipFree A");
    cuda_check_error(hipFree(d_b), "hipFree B");
    cuda_check_error(hipFree(d_result), "hipFree result");

    return result;
}

// Helper function to launch matrix_product_tiled kernels
template<typename T>
py::array_t<T> matrix_product_tiled_cuda_impl(
    const py::array_t<T>& a,
    const py::array_t<T>& b
) {
    // Validate input arrays
    auto a_buf = a.request();
    auto b_buf = b.request();

    if (a_buf.ndim != 2 || b_buf.ndim != 2) {
        throw std::invalid_argument("Input arrays must be 2-dimensional");
    }

    if (!a.flags() & py::array::c_style) {
        throw std::invalid_argument("Array a must be C-contiguous");
    }

    if (!b.flags() & py::array::c_style) {
        throw std::invalid_argument("Array b must be C-contiguous");
    }

    // Get matrix dimensions
    long m = a_buf.shape[0];  // rows of A
    long n = a_buf.shape[1];  // cols of A, rows of B
    long k = b_buf.shape[1];  // cols of B

    if (b_buf.shape[0] != n) {
        throw std::invalid_argument("Matrix dimensions incompatible for multiplication");
    }

    // Create output array
    auto result = py::array_t<T>({m, k});
    auto result_buf = result.request();

    // Get data pointers
    const T* a_ptr = static_cast<const T*>(a_buf.ptr);
    const T* b_ptr = static_cast<const T*>(b_buf.ptr);
    T* result_ptr = static_cast<T*>(result_buf.ptr);

    // Create kernel specification
    Matrix_product_tiled_spec spec(
        "float",  // Type string (will be overridden by template parameter)
        m, n, k
    );

    // Create kernel instance
    Matrix_product_tiled_kernel<T> kernel(spec);

    // Allocate GPU memory
    T* d_a = nullptr;
    T* d_b = nullptr;
    T* d_result = nullptr;

    size_t size_a = m * n * sizeof(T);
    size_t size_b = n * k * sizeof(T);
    size_t size_result = m * k * sizeof(T);

    cuda_check_error(hipMalloc(&d_a, size_a), "hipMalloc for matrix A");
    cuda_check_error(hipMalloc(&d_b, size_b), "hipMalloc for matrix B");
    cuda_check_error(hipMalloc(&d_result, size_result), "hipMalloc for result matrix");

    try {
        // Copy data to device
        cuda_check_error(hipMemcpy(d_a, a_ptr, size_a, hipMemcpyHostToDevice), "hipMemcpy A to device");
        cuda_check_error(hipMemcpy(d_b, b_ptr, size_b, hipMemcpyHostToDevice), "hipMemcpy B to device");

        // Create CUDA stream
        hipStream_t stream;
        cuda_check_error(hipStreamCreate(&stream), "hipStreamCreate");

        // Run kernel
        kernel.run_device_kernel(d_a, d_b, d_result, nullptr, stream);

        // Wait for completion
        cuda_check_error(hipStreamSynchronize(stream), "hipStreamSynchronize");

        // Copy result back to host
        cuda_check_error(hipMemcpy(result_ptr, d_result, size_result, hipMemcpyDeviceToHost), "hipMemcpy result to host");

        // Cleanup stream
        cuda_check_error(hipStreamDestroy(stream), "hipStreamDestroy");

    } catch (...) {
        // Cleanup on error
        if (d_a) hipFree(d_a);
        if (d_b) hipFree(d_b);
        if (d_result) hipFree(d_result);
        throw;
    }

    // Cleanup GPU memory
    cuda_check_error(hipFree(d_a), "hipFree A");
    cuda_check_error(hipFree(d_b), "hipFree B");
    cuda_check_error(hipFree(d_result), "hipFree result");

    return result;
}

// Helper function to launch matrix_transpose_striped kernels
template<typename T>
py::array_t<T> matrix_transpose_striped_cuda_impl(const py::array_t<T>& a) {
    // Validate input array
    auto a_buf = a.request();

    if (a_buf.ndim != 2) {
        throw std::invalid_argument("Input array must be 2-dimensional");
    }

    if (!a.flags() & py::array::c_style) {
        throw std::invalid_argument("Array must be C-contiguous");
    }

    // Get matrix dimensions
    long m = a_buf.shape[0];  // rows of A
    long n = a_buf.shape[1];  // cols of A

    // Create output array (transposed dimensions)
    auto result = py::array_t<T>({n, m});
    auto result_buf = result.request();

    // Get data pointers
    const T* a_ptr = static_cast<const T*>(a_buf.ptr);
    T* result_ptr = static_cast<T*>(result_buf.ptr);

    // Create kernel specification
    Matrix_transpose_striped_spec spec(
        "float",  // Type string (will be overridden by template parameter)
        m, n,
        32        // Default block dimension
    );

    // Create kernel instance
    Matrix_transpose_striped_kernel<T> kernel(spec);

    // Allocate GPU memory
    T* d_a = nullptr;
    T* d_result = nullptr;

    size_t size_a = m * n * sizeof(T);
    size_t size_result = n * m * sizeof(T);

    cuda_check_error(hipMalloc(&d_a, size_a), "hipMalloc for matrix A");
    cuda_check_error(hipMalloc(&d_result, size_result), "hipMalloc for result matrix");

    try {
        // Copy data to device
        cuda_check_error(hipMemcpy(d_a, a_ptr, size_a, hipMemcpyHostToDevice), "hipMemcpy A to device");

        // Create CUDA stream
        hipStream_t stream;
        cuda_check_error(hipStreamCreate(&stream), "hipStreamCreate");

        // Run kernel
        kernel.run_device_kernel(d_a, d_result, nullptr, stream);

        // Wait for completion
        cuda_check_error(hipStreamSynchronize(stream), "hipStreamSynchronize");

        // Copy result back to host
        cuda_check_error(hipMemcpy(result_ptr, d_result, size_result, hipMemcpyDeviceToHost), "hipMemcpy result to host");

        // Cleanup stream
        cuda_check_error(hipStreamDestroy(stream), "hipStreamDestroy");

    } catch (...) {
        // Cleanup on error
        if (d_a) hipFree(d_a);
        if (d_result) hipFree(d_result);
        throw;
    }

    // Cleanup GPU memory
    cuda_check_error(hipFree(d_a), "hipFree A");
    cuda_check_error(hipFree(d_result), "hipFree result");

    return result;
}

// High-level dispatch function that determines type at runtime
py::object matrix_product_naive_dispatch(py::array a, py::array b) {
    if (!a.dtype().is(b.dtype())) {
        throw std::invalid_argument("Input arrays must have the same dtype");
    }

    // TODO: Handle float16 properly - commented out until we figure out the right approach
    // if (a.dtype().is(py::dtype("float16"))) {
    //     return matrix_product_naive_cuda_impl<__half>(a.cast<py::array_t<__half>>(), b.cast<py::array_t<__half>>());
    // } else
    if (a.dtype().is(py::dtype::of<float>())) {
        return matrix_product_naive_cuda_impl<float>(a.cast<py::array_t<float>>(), b.cast<py::array_t<float>>());
    } else if (a.dtype().is(py::dtype::of<double>())) {
        return matrix_product_naive_cuda_impl<double>(a.cast<py::array_t<double>>(), b.cast<py::array_t<double>>());
    } else if (a.dtype().is(py::dtype::of<std::int8_t>())) {
        return matrix_product_naive_cuda_impl<std::int8_t>(a.cast<py::array_t<std::int8_t>>(), b.cast<py::array_t<std::int8_t>>());
    } else if (a.dtype().is(py::dtype::of<std::int16_t>())) {
        return matrix_product_naive_cuda_impl<std::int16_t>(a.cast<py::array_t<std::int16_t>>(), b.cast<py::array_t<std::int16_t>>());
    } else if (a.dtype().is(py::dtype::of<std::int32_t>())) {
        return matrix_product_naive_cuda_impl<std::int32_t>(a.cast<py::array_t<std::int32_t>>(), b.cast<py::array_t<std::int32_t>>());
    } else if (a.dtype().is(py::dtype::of<std::int64_t>())) {
        return matrix_product_naive_cuda_impl<std::int64_t>(a.cast<py::array_t<std::int64_t>>(), b.cast<py::array_t<std::int64_t>>());
    } else if (a.dtype().is(py::dtype::of<std::uint8_t>())) {
        return matrix_product_naive_cuda_impl<std::uint8_t>(a.cast<py::array_t<std::uint8_t>>(), b.cast<py::array_t<std::uint8_t>>());
    } else if (a.dtype().is(py::dtype::of<std::uint16_t>())) {
        return matrix_product_naive_cuda_impl<std::uint16_t>(a.cast<py::array_t<std::uint16_t>>(), b.cast<py::array_t<std::uint16_t>>());
    } else if (a.dtype().is(py::dtype::of<std::uint32_t>())) {
        return matrix_product_naive_cuda_impl<std::uint32_t>(a.cast<py::array_t<std::uint32_t>>(), b.cast<py::array_t<std::uint32_t>>());
    } else if (a.dtype().is(py::dtype::of<std::uint64_t>())) {
        return matrix_product_naive_cuda_impl<std::uint64_t>(a.cast<py::array_t<std::uint64_t>>(), b.cast<py::array_t<std::uint64_t>>());
    } else {
        throw std::invalid_argument("Unsupported dtype: " + py::str(a.dtype()).cast<std::string>());
    }
}

// High-level dispatch functions for additional matrix operations
py::object matrix_product_tiled_dispatch(py::array a, py::array b) {
    if (!a.dtype().is(b.dtype())) {
        throw std::invalid_argument("Input arrays must have the same dtype");
    }

    if (a.dtype().is(py::dtype::of<float>())) {
        return matrix_product_tiled_cuda_impl<float>(a.cast<py::array_t<float>>(), b.cast<py::array_t<float>>());
    } else if (a.dtype().is(py::dtype::of<double>())) {
        return matrix_product_tiled_cuda_impl<double>(a.cast<py::array_t<double>>(), b.cast<py::array_t<double>>());
    } else if (a.dtype().is(py::dtype::of<std::int8_t>())) {
        return matrix_product_tiled_cuda_impl<std::int8_t>(a.cast<py::array_t<std::int8_t>>(), b.cast<py::array_t<std::int8_t>>());
    } else if (a.dtype().is(py::dtype::of<std::int16_t>())) {
        return matrix_product_tiled_cuda_impl<std::int16_t>(a.cast<py::array_t<std::int16_t>>(), b.cast<py::array_t<std::int16_t>>());
    } else if (a.dtype().is(py::dtype::of<std::int32_t>())) {
        return matrix_product_tiled_cuda_impl<std::int32_t>(a.cast<py::array_t<std::int32_t>>(), b.cast<py::array_t<std::int32_t>>());
    } else if (a.dtype().is(py::dtype::of<std::int64_t>())) {
        return matrix_product_tiled_cuda_impl<std::int64_t>(a.cast<py::array_t<std::int64_t>>(), b.cast<py::array_t<std::int64_t>>());
    } else if (a.dtype().is(py::dtype::of<std::uint8_t>())) {
        return matrix_product_tiled_cuda_impl<std::uint8_t>(a.cast<py::array_t<std::uint8_t>>(), b.cast<py::array_t<std::uint8_t>>());
    } else if (a.dtype().is(py::dtype::of<std::uint16_t>())) {
        return matrix_product_tiled_cuda_impl<std::uint16_t>(a.cast<py::array_t<std::uint16_t>>(), b.cast<py::array_t<std::uint16_t>>());
    } else if (a.dtype().is(py::dtype::of<std::uint32_t>())) {
        return matrix_product_tiled_cuda_impl<std::uint32_t>(a.cast<py::array_t<std::uint32_t>>(), b.cast<py::array_t<std::uint32_t>>());
    } else if (a.dtype().is(py::dtype::of<std::uint64_t>())) {
        return matrix_product_tiled_cuda_impl<std::uint64_t>(a.cast<py::array_t<std::uint64_t>>(), b.cast<py::array_t<std::uint64_t>>());
    } else {
        throw std::invalid_argument("Unsupported dtype for tiled matrix product: " + py::str(a.dtype()).cast<std::string>());
    }
}

py::object matrix_transpose_striped_dispatch(py::array a) {
    if (a.dtype().is(py::dtype::of<float>())) {
        return matrix_transpose_striped_cuda_impl<float>(a.cast<py::array_t<float>>());
    } else if (a.dtype().is(py::dtype::of<double>())) {
        return matrix_transpose_striped_cuda_impl<double>(a.cast<py::array_t<double>>());
    } else if (a.dtype().is(py::dtype::of<std::int8_t>())) {
        return matrix_transpose_striped_cuda_impl<std::int8_t>(a.cast<py::array_t<std::int8_t>>());
    } else if (a.dtype().is(py::dtype::of<std::int16_t>())) {
        return matrix_transpose_striped_cuda_impl<std::int16_t>(a.cast<py::array_t<std::int16_t>>());
    } else if (a.dtype().is(py::dtype::of<std::int32_t>())) {
        return matrix_transpose_striped_cuda_impl<std::int32_t>(a.cast<py::array_t<std::int32_t>>());
    } else if (a.dtype().is(py::dtype::of<std::int64_t>())) {
        return matrix_transpose_striped_cuda_impl<std::int64_t>(a.cast<py::array_t<std::int64_t>>());
    } else if (a.dtype().is(py::dtype::of<std::uint8_t>())) {
        return matrix_transpose_striped_cuda_impl<std::uint8_t>(a.cast<py::array_t<std::uint8_t>>());
    } else if (a.dtype().is(py::dtype::of<std::uint16_t>())) {
        return matrix_transpose_striped_cuda_impl<std::uint16_t>(a.cast<py::array_t<std::uint16_t>>());
    } else if (a.dtype().is(py::dtype::of<std::uint32_t>())) {
        return matrix_transpose_striped_cuda_impl<std::uint32_t>(a.cast<py::array_t<std::uint32_t>>());
    } else if (a.dtype().is(py::dtype::of<std::uint64_t>())) {
        return matrix_transpose_striped_cuda_impl<std::uint64_t>(a.cast<py::array_t<std::uint64_t>>());
    } else {
        throw std::invalid_argument("Unsupported dtype: " + py::str(a.dtype()).cast<std::string>());
    }
}

// Python module definition
PYBIND11_MODULE(_matrix_ops_cuda, m) {
    m.doc() = "CUDA matrix operations for py-gpu-algos";

    // Low-level type-specific functions
    // TODO: Handle float16 properly - commented out until we figure out the right approach
    // m.def("matrix_product_naive_float16", &matrix_product_naive_cuda_impl<__half>,
    //       "Matrix multiplication (naive algorithm) for float16",
    //       py::arg("a"), py::arg("b"));

    m.def("matrix_product_naive_float32", &matrix_product_naive_cuda_impl<float>,
          "Matrix multiplication (naive algorithm) for float32",
          py::arg("a"), py::arg("b"));

    m.def("matrix_product_naive_float64", &matrix_product_naive_cuda_impl<double>,
          "Matrix multiplication (naive algorithm) for float64",
          py::arg("a"), py::arg("b"));

    m.def("matrix_product_naive_int8", &matrix_product_naive_cuda_impl<std::int8_t>,
          "Matrix multiplication (naive algorithm) for int8",
          py::arg("a"), py::arg("b"));

    m.def("matrix_product_naive_int16", &matrix_product_naive_cuda_impl<std::int16_t>,
          "Matrix multiplication (naive algorithm) for int16",
          py::arg("a"), py::arg("b"));

    m.def("matrix_product_naive_int32", &matrix_product_naive_cuda_impl<std::int32_t>,
          "Matrix multiplication (naive algorithm) for int32",
          py::arg("a"), py::arg("b"));

    m.def("matrix_product_naive_int64", &matrix_product_naive_cuda_impl<std::int64_t>,
          "Matrix multiplication (naive algorithm) for int64",
          py::arg("a"), py::arg("b"));

    m.def("matrix_product_naive_uint8", &matrix_product_naive_cuda_impl<std::uint8_t>,
          "Matrix multiplication (naive algorithm) for uint8",
          py::arg("a"), py::arg("b"));

    m.def("matrix_product_naive_uint16", &matrix_product_naive_cuda_impl<std::uint16_t>,
          "Matrix multiplication (naive algorithm) for uint16",
          py::arg("a"), py::arg("b"));

    m.def("matrix_product_naive_uint32", &matrix_product_naive_cuda_impl<std::uint32_t>,
          "Matrix multiplication (naive algorithm) for uint32",
          py::arg("a"), py::arg("b"));

    m.def("matrix_product_naive_uint64", &matrix_product_naive_cuda_impl<std::uint64_t>,
          "Matrix multiplication (naive algorithm) for uint64",
          py::arg("a"), py::arg("b"));

    // High-level dispatch function
    m.def("matrix_product_naive", &matrix_product_naive_dispatch,
          "Matrix multiplication (naive algorithm) with automatic type dispatch",
          py::arg("a"), py::arg("b"));

    // Additional matrix operations - tiled
    m.def("matrix_product_tiled_float32", &matrix_product_tiled_cuda_impl<float>,
          "Matrix multiplication (tiled algorithm) for float32",
          py::arg("a"), py::arg("b"));
    m.def("matrix_product_tiled_float64", &matrix_product_tiled_cuda_impl<double>,
          "Matrix multiplication (tiled algorithm) for float64",
          py::arg("a"), py::arg("b"));
    m.def("matrix_product_tiled_int8", &matrix_product_tiled_cuda_impl<std::int8_t>,
          "Matrix multiplication (tiled algorithm) for int8",
          py::arg("a"), py::arg("b"));
    m.def("matrix_product_tiled_int16", &matrix_product_tiled_cuda_impl<std::int16_t>,
          "Matrix multiplication (tiled algorithm) for int16",
          py::arg("a"), py::arg("b"));
    m.def("matrix_product_tiled_int32", &matrix_product_tiled_cuda_impl<std::int32_t>,
          "Matrix multiplication (tiled algorithm) for int32",
          py::arg("a"), py::arg("b"));
    m.def("matrix_product_tiled_int64", &matrix_product_tiled_cuda_impl<std::int64_t>,
          "Matrix multiplication (tiled algorithm) for int64",
          py::arg("a"), py::arg("b"));
    m.def("matrix_product_tiled_uint8", &matrix_product_tiled_cuda_impl<std::uint8_t>,
          "Matrix multiplication (tiled algorithm) for uint8",
          py::arg("a"), py::arg("b"));
    m.def("matrix_product_tiled_uint16", &matrix_product_tiled_cuda_impl<std::uint16_t>,
          "Matrix multiplication (tiled algorithm) for uint16",
          py::arg("a"), py::arg("b"));
    m.def("matrix_product_tiled_uint32", &matrix_product_tiled_cuda_impl<std::uint32_t>,
          "Matrix multiplication (tiled algorithm) for uint32",
          py::arg("a"), py::arg("b"));
    m.def("matrix_product_tiled_uint64", &matrix_product_tiled_cuda_impl<std::uint64_t>,
          "Matrix multiplication (tiled algorithm) for uint64",
          py::arg("a"), py::arg("b"));
    m.def("matrix_product_tiled", &matrix_product_tiled_dispatch,
          "Matrix multiplication (tiled algorithm) with automatic type dispatch",
          py::arg("a"), py::arg("b"));

    // Matrix transpose operations - striped
    m.def("matrix_transpose_striped_float32", &matrix_transpose_striped_cuda_impl<float>,
          "Matrix transpose (striped algorithm) for float32", py::arg("a"));
    m.def("matrix_transpose_striped_float64", &matrix_transpose_striped_cuda_impl<double>,
          "Matrix transpose (striped algorithm) for float64", py::arg("a"));
    m.def("matrix_transpose_striped_int8", &matrix_transpose_striped_cuda_impl<std::int8_t>,
          "Matrix transpose (striped algorithm) for int8", py::arg("a"));
    m.def("matrix_transpose_striped_int16", &matrix_transpose_striped_cuda_impl<std::int16_t>,
          "Matrix transpose (striped algorithm) for int16", py::arg("a"));
    m.def("matrix_transpose_striped_int32", &matrix_transpose_striped_cuda_impl<std::int32_t>,
          "Matrix transpose (striped algorithm) for int32", py::arg("a"));
    m.def("matrix_transpose_striped_int64", &matrix_transpose_striped_cuda_impl<std::int64_t>,
          "Matrix transpose (striped algorithm) for int64", py::arg("a"));
    m.def("matrix_transpose_striped_uint8", &matrix_transpose_striped_cuda_impl<std::uint8_t>,
          "Matrix transpose (striped algorithm) for uint8", py::arg("a"));
    m.def("matrix_transpose_striped_uint16", &matrix_transpose_striped_cuda_impl<std::uint16_t>,
          "Matrix transpose (striped algorithm) for uint16", py::arg("a"));
    m.def("matrix_transpose_striped_uint32", &matrix_transpose_striped_cuda_impl<std::uint32_t>,
          "Matrix transpose (striped algorithm) for uint32", py::arg("a"));
    m.def("matrix_transpose_striped_uint64", &matrix_transpose_striped_cuda_impl<std::uint64_t>,
          "Matrix transpose (striped algorithm) for uint64", py::arg("a"));
    m.def("matrix_transpose_striped", &matrix_transpose_striped_dispatch,
          "Matrix transpose (striped algorithm) with automatic type dispatch", py::arg("a"));
}

// #endif // WITH_CUDA
