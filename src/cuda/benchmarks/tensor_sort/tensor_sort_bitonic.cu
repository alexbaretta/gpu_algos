#include "hip/hip_runtime.h"
// Copyright (c) 2025 Alessandro Baretta
// All rights reserved.

// source path: src/benchmarks/cuda/tensor_sort/tensor_sort_bitonic.cu

#include <cxxopts.hpp>

#include "common/benchmark_options.hpp"
#include "cuda/benchmark/benchmark_tensor3d_1inout.cuh"

#include "cuda/kernels/sort/tensor_sort_bitonic.cuh"

int main(int argc, char** argv) {
    cxxopts::Options options("tensor_sort_bitonic", "Tensor sort (bitonic algorithm)");
    add_benchmark_options(options);
    tensor3d_sort_bitonic_spec::add_kernel_spec_options(options);

    try {
        cxxopts::ParseResult options_parsed = options.parse(argc, argv);

        tensor3d_sort_bitonic_spec spec = tensor3d_sort_bitonic_spec::make(options_parsed);

        if (spec.type_ == "half") {
            return Benchmark_Tensor3D_1Inout<tensor3d_sort_bitonic_kernel<__half>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "single" || spec.type_ == "float") {
            return Benchmark_Tensor3D_1Inout<tensor3d_sort_bitonic_kernel<float>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "double") {
            return Benchmark_Tensor3D_1Inout<tensor3d_sort_bitonic_kernel<double>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "int8") {
            return Benchmark_Tensor3D_1Inout<tensor3d_sort_bitonic_kernel<std::int8_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "int16") {
            return Benchmark_Tensor3D_1Inout<tensor3d_sort_bitonic_kernel<std::int16_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "int32") {
            return Benchmark_Tensor3D_1Inout<tensor3d_sort_bitonic_kernel<std::int32_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "int64") {
            return Benchmark_Tensor3D_1Inout<tensor3d_sort_bitonic_kernel<std::int64_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "uint8") {
            return Benchmark_Tensor3D_1Inout<tensor3d_sort_bitonic_kernel<std::uint8_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "uint16") {
            return Benchmark_Tensor3D_1Inout<tensor3d_sort_bitonic_kernel<std::uint16_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "uint32") {
            return Benchmark_Tensor3D_1Inout<tensor3d_sort_bitonic_kernel<std::uint32_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "uint64") {
            return Benchmark_Tensor3D_1Inout<tensor3d_sort_bitonic_kernel<std::uint64_t>>(spec, options, options_parsed).run();
        } else {
            throw cxxopts::exceptions::exception("Invalid type: " + spec.type_);
        }
    } catch (const cxxopts::exceptions::exception& e) {
       std::cerr << "Error parsing options: " << e.what() << std::endl;
       std::cout << options.help() << std::endl;
       return 1;
    }
}
