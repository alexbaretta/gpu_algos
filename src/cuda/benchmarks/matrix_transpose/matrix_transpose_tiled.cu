#include "hip/hip_runtime.h"
/*
    Copyright (c) 2025 Alessandro Baretta <alex@baretta.com>

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/


// source path: src/benchmarks/matrix_transpose/matrix_transpose_tiled.cu

#include <cxxopts.hpp>

#include "common/benchmark_options.hpp"
#include "cuda/benchmark/benchmark_matrix_1in_1out.cuh"

#include "cuda/kernels/matrix_transpose/matrix_transpose_tiled.cuh"

int main(int argc, char** argv) {
    cxxopts::Options options("matrix_transpose_tiled", "Matrix transpose (tiled algorithm)");
    add_benchmark_options(options);
    Matrix_transpose_tiled_spec::add_kernel_spec_options(options);

    try {
        cxxopts::ParseResult options_parsed = options.parse(argc, argv);

        Matrix_transpose_tiled_spec spec = Matrix_transpose_tiled_spec::make(options_parsed);

        if (spec.type_ == "half") {
            return Benchmark_Matrix_1In_1Out<Matrix_transpose_tiled_kernel<__half>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "single" || spec.type_ == "float") {
            return Benchmark_Matrix_1In_1Out<Matrix_transpose_tiled_kernel<float>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "double") {
            return Benchmark_Matrix_1In_1Out<Matrix_transpose_tiled_kernel<double>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "int8") {
            return Benchmark_Matrix_1In_1Out<Matrix_transpose_tiled_kernel<std::int8_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "int16") {
            return Benchmark_Matrix_1In_1Out<Matrix_transpose_tiled_kernel<std::int16_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "int32") {
            return Benchmark_Matrix_1In_1Out<Matrix_transpose_tiled_kernel<std::int32_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "int64") {
            return Benchmark_Matrix_1In_1Out<Matrix_transpose_tiled_kernel<std::int64_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "uint8") {
            return Benchmark_Matrix_1In_1Out<Matrix_transpose_tiled_kernel<std::uint8_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "uint16") {
            return Benchmark_Matrix_1In_1Out<Matrix_transpose_tiled_kernel<std::uint16_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "uint32") {
            return Benchmark_Matrix_1In_1Out<Matrix_transpose_tiled_kernel<std::uint32_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "uint64") {
            return Benchmark_Matrix_1In_1Out<Matrix_transpose_tiled_kernel<std::uint64_t>>(spec, options, options_parsed).run();
        } else {
            throw cxxopts::exceptions::exception("Invalid type: " + spec.type_);
        }
    } catch (const cxxopts::exceptions::exception& e) {
       std::cerr << "Error parsing options: " << e.what() << std::endl;
       std::cout << options.help() << std::endl;
       return 1;
    }
}
