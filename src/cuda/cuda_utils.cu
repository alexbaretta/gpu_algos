#include "hip/hip_runtime.h"
/*
    Copyright (c) 2025 Alessandro Baretta <alex@baretta.com>

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/


// source path: src/cuda/cuda_utils.cu

#include <chrono>

#include "cuda/cuda_utils.cuh"

void report_completion_time_callback(hipStream_t stream, hipError_t status, void* userData) {
    // This is a CUDA callback: it may not call cuda functions
    // It runs in a separate thread, so it may not write to iostreams
    auto& time = *static_cast<std::chrono::high_resolution_clock::time_point*>(userData);
    time = std::chrono::high_resolution_clock::now();
}

hipDeviceProp_t get_device_prop(const int device_id) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);
    return prop;
}
hipDeviceProp_t get_default_device_prop() {
    int device_id;
    hipGetDevice(&device_id);
    return get_device_prop(device_id);
}

// Template specializations for __half type
template <>
__host__ __device__ __half cuda_max<__half>(__half a, __half b) {
    return __hmax(a, b);
}

template <>
__host__ __device__ __half cuda_min<__half>(__half a, __half b) {
    return __hmin(a, b);
}

template <>
__host__ __device__ __half device_nan<__half>() {
    return __ushort_as_half(0x7e00);
}
