#include "hip/hip_runtime.h"
// Copyright (c) 2025 Alessandro Baretta
// All rights reserved.

// source path: src/benchmarks/cuda/gradient/gradient_leastsquares_elasticnet_optimizer.cu

#include <cxxopts.hpp>

#include "common/benchmark_options.hpp"
#include "common/benchmark/benchmark_matrix_3in_1out.hpp"

#include "cuda/kernels/gradient/gradient_leastsquares_elasticnet_optimizer.hpp"

int main(int argc, char** argv) {
    cxxopts::Options options("gradient_leastsquares_elasticnet_optimizer", "Gradient descent optimization with ElasticNet regularization and line sear.hpp");
    add_benchmark_options(options);
    Gradient_leastsquares_elasticnet_optimizer_spec::add_kernel_spec_options(options);

    try {
        cxxopts::ParseResult options_parsed = options.parse(argc, argv);

        Gradient_leastsquares_elasticnet_optimizer_spec spec = Gradient_leastsquares_elasticnet_optimizer_spec::make(options_parsed);

        if (spec.type_ == "half") {
            return Benchmark_Matrix_3In_1Out<Gradient_leastsquares_elasticnet_optimizer_kernel<__half>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "single" || spec.type_ == "float") {
            return Benchmark_Matrix_3In_1Out<Gradient_leastsquares_elasticnet_optimizer_kernel<float>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "double") {
            return Benchmark_Matrix_3In_1Out<Gradient_leastsquares_elasticnet_optimizer_kernel<double>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "int8") {
            return Benchmark_Matrix_3In_1Out<Gradient_leastsquares_elasticnet_optimizer_kernel<std::int8_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "int16") {
            return Benchmark_Matrix_3In_1Out<Gradient_leastsquares_elasticnet_optimizer_kernel<std::int16_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "int32") {
            return Benchmark_Matrix_3In_1Out<Gradient_leastsquares_elasticnet_optimizer_kernel<std::int32_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "int64") {
            return Benchmark_Matrix_3In_1Out<Gradient_leastsquares_elasticnet_optimizer_kernel<std::int64_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "uint8") {
            return Benchmark_Matrix_3In_1Out<Gradient_leastsquares_elasticnet_optimizer_kernel<std::uint8_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "uint16") {
            return Benchmark_Matrix_3In_1Out<Gradient_leastsquares_elasticnet_optimizer_kernel<std::uint16_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "uint32") {
            return Benchmark_Matrix_3In_1Out<Gradient_leastsquares_elasticnet_optimizer_kernel<std::uint32_t>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "uint64") {
            return Benchmark_Matrix_3In_1Out<Gradient_leastsquares_elasticnet_optimizer_kernel<std::uint64_t>>(spec, options, options_parsed).run();
        } else {
            throw cxxopts::exceptions::exception("Invalid type: " + spec.type_);
        }
    } catch (const cxxopts::exceptions::exception& e) {
       std::cerr << "Error parsing options: " << e.what() << std::endl;
       std::cout << options.help() << std::endl;
       return 1;
    }
}
