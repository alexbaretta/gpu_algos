#include "hip/hip_runtime.h"
// Copyright (c) 2025 Alessandro Baretta
// All rights reserved.

// source path: src/tools/matrix_transpose/matrix_transpose_tiled.cu

#include <cxxopts.hpp>

#include "common/benchmark.h"

#include "cuda/kernels/matrix_transpose/matrix_transpose_tiled.h"

int main(int argc, char** argv) {
    cxxopts::Options options("matrix_transpose_tiled", "tiled matrix multiplication");
    add_benchmark_options(options);
    Matrix_transpose_tiled_spec::add_kernel_spec_options(options);

    try {
        cxxopts::ParseResult options_parsed = options.parse(argc, argv);

        Matrix_transpose_tiled_spec spec = Matrix_transpose_tiled_spec::make(options_parsed);

        if (spec.type_ == "half") {
            return Benchmark_1In_1Out<Matrix_transpose_tiled_kernel<__half>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "single" || spec.type_ == "float") {
            return Benchmark_1In_1Out<Matrix_transpose_tiled_kernel<float>>(spec, options, options_parsed).run();
        } else if (spec.type_ == "double") {
            return Benchmark_1In_1Out<Matrix_transpose_tiled_kernel<double>>(spec, options, options_parsed).run();
        }
    } catch (const cxxopts::exceptions::exception& e) {
       std::cerr << "Error parsing options: " << e.what() << std::endl;
       std::cout << options.help() << std::endl;
       return 1;
    }
}
