#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <iostream>
#include <random>
#include <ratio>
#include <vector>

#include <hip/hip_runtime.h>
#include <cxxopts.hpp>
#include <Eigen/Core>

#include "common/cuda/check_errors.h"
#include "common/cuda/cuda_utils.h"
#include "common/eigen/algorithms.h"

// Default matrix dimensions
constexpr int DEFAULT_M = 1000;    // Rows of first matrix
constexpr int DEFAULT_N = 10000;   // Columns of first matrix / Rows of second matrix
constexpr int DEFAULT_GPU_MEM = 16; // GPU memory size in GB
constexpr unsigned int NULL_FLAGS = 0;

// CUDA kernel for matrix multiplication
__global__ void matrixMultiply(const float* A, const float* B, float* C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            sum += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}

// Function to initialize matrix with random values
void initialize_matrix(std::vector<float>& matrix, int rows, int cols) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(0.0f, 1.0f);

    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] = dis(gen);
    }
}


int main(int argc, char** argv) {
    // Parse command line arguments
    cxxopts::Options options("matrix_multiply", "CUDA Matrix Multiplication");
    options.add_options()
        ("nrows", "Number of rows in first matrix", cxxopts::value<int>()->default_value(std::to_string(DEFAULT_M)))
        ("ncols", "Number of columns in first matrix", cxxopts::value<int>()->default_value(std::to_string(DEFAULT_N)))
        ("gpumem", "GPU memory size", cxxopts::value<int>()->default_value(std::to_string(DEFAULT_GPU_MEM)))
        ("h,help", "Print usage");

    try {
        auto result = options.parse(argc, argv);

        if (result.count("help")) {
            std::cout << options.help() << std::endl;
            return 0;
        }

        // Get matrix dimensions from command line or use defaults
        const int M = result["nrows"].as<int>();
        const int N = result["ncols"].as<int>();
        const int size = M * N;
        const int gpu_mem = result["gpumem"].as<int>();
        constexpr float GB = 1024.0f * 1024.0f * 1024.0f;
        const float matrix_size_gb = size * sizeof(float) / GB;
        const float mem_gb = matrix_size_gb * 3;
        std::cout
            << "Matrix dimensions: " << M << "x" << N << " * " << N << "x" << M << "\n"
            << "Matrix size      : " << matrix_size_gb << " GB\n"
            << "Required mem     : " << mem_gb << " GB"
            << std::endl;
        if (mem_gb > gpu_mem) {
            std::cerr << "[ERROR] GPU memory size is less than the matrix size" << std::endl;
            return 1;
        }


        std::cout << "SETUP:" << std::endl;
        const auto setup_tp0 = std::chrono::high_resolution_clock::now();

        std::cout << "  - Allocating memory: ";
        std::vector<float> h_A(size, 0.0f);
        std::vector<float> h_B(size, 0.0f);
        std::vector<float> h_C(size, 0.0f);
        const auto setup_tp1 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> setup_dt1 = setup_tp1 - setup_tp0;
        std::cout << setup_dt1.count() << " ms (" << setup_dt1.count() << " ms total)" << std::endl;

        std::cout << "  - Initializing matrices: ";
        initialize_matrix(h_A, M, N);
        initialize_matrix(h_B, N, M);
        const auto setup_tp2 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> setup_step_dt2 = setup_tp2 - setup_tp1;
        std::chrono::duration<double, std::milli> setup_total_dt2 = setup_tp2 - setup_tp0;
        std::cout << setup_step_dt2.count() << " ms (" << setup_total_dt2.count() << " ms total)" << std::endl;

        std::cout << "  - Creating GPU streams: ";
        hipStream_t stream;
        cuda_check_error(hipStreamCreate(&stream), "hipStreamCreate");
        const auto setup_tp3 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> setup_step_dt3 = setup_tp3 - setup_tp2;
        std::chrono::duration<double, std::milli> setup_total_dt3 = setup_tp3 - setup_tp0;
        std::cout << setup_step_dt3.count() << " ms (" << setup_total_dt3.count() << " ms total)" << std::endl;

        std::cout << "  - Creating GPU events: ";
        hipEvent_t e0, e1, e2, e3, e4, e5;
        cuda_check_error(hipEventCreate(&e0), "hipEventCreate");
        cuda_check_error(hipEventCreate(&e1), "hipEventCreate");
        cuda_check_error(hipEventCreate(&e2), "hipEventCreate");
        cuda_check_error(hipEventCreate(&e3), "hipEventCreate");
        cuda_check_error(hipEventCreate(&e4), "hipEventCreate");
        cuda_check_error(hipEventCreate(&e5), "hipEventCreate");
        const auto setup_tp4 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> setup_step_dt4 = setup_tp4 - setup_tp3;
        std::chrono::duration<double, std::milli> setup_total_dt4 = setup_tp4 - setup_tp0;
        std::cout << setup_step_dt4.count() << " ms (" << setup_total_dt4.count() << " ms total)" << std::endl;


        std::cout << "KERNEL:" << std::endl;
        const auto gpu_tp0 = std::chrono::high_resolution_clock::now();
        cuda_check_error(hipEventRecord(e0, stream), "hipEventRecord");

        const auto gpu_step_1 = "Allocate device memory";
        float *d_A, *d_B, *d_C;
        cuda_check_error(hipMallocAsync(&d_A, size * sizeof(float), stream), "hipMallocAsync");
        cuda_check_error(hipMallocAsync(&d_B, size * sizeof(float), stream), "hipMallocAsync");
        cuda_check_error(hipMallocAsync(&d_C, size * sizeof(float), stream), "hipMallocAsync");
        cuda_check_error(hipEventRecord(e1, stream), "hipEventRecord");
        std::chrono::high_resolution_clock::time_point gpu_tp1{};
        hipStreamAddCallback(stream, report_completion_time_callback, &gpu_tp1, NULL_FLAGS);

        const auto gpu_step_2 = "Copy data to device";
        cuda_check_error(hipMemcpyAsync(d_A, h_A.data(), size * sizeof(float), hipMemcpyHostToDevice, stream), "hipMemcpyAsync");
        cuda_check_error(hipMemcpyAsync(d_B, h_B.data(), size * sizeof(float), hipMemcpyHostToDevice, stream), "hipMemcpyAsync");
        cuda_check_error(hipEventRecord(e2, stream), "hipEventRecord");
        std::chrono::high_resolution_clock::time_point gpu_tp2{};
        hipStreamAddCallback(stream, report_completion_time_callback, &gpu_tp2, NULL_FLAGS);

        const auto gpu_step_3 = "Compute kernel";
        dim3 blockDim(16, 16);
        dim3 gridDim((M + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);
        matrixMultiply<<<gridDim, blockDim, 0, stream>>>(d_A, d_B, d_C, M, N, M);
        cuda_check_error(hipEventRecord(e3, stream), "hipEventRecord");
        std::chrono::high_resolution_clock::time_point gpu_tp3{};
        cuda_check_error(hipStreamAddCallback(stream, report_completion_time_callback, &gpu_tp3, NULL_FLAGS), "hipStreamAddCallback");

        const auto gpu_step_4 = "Copy result back to host";
        cuda_check_error(hipMemcpyAsync(h_C.data(), d_C, size * sizeof(float), hipMemcpyDeviceToHost, stream), "hipMemcpyAsync");
        cuda_check_error(hipEventRecord(e4, stream), "hipEventRecord");
        std::chrono::high_resolution_clock::time_point gpu_tp4{};
        cuda_check_error(hipStreamAddCallback(stream, report_completion_time_callback, &gpu_tp4, NULL_FLAGS), "hipStreamAddCallback");

        const auto gpu_step_5 = "Free device memory";
        cuda_check_error(hipFreeAsync(d_A, stream), "hipFreeAsync");
        cuda_check_error(hipFreeAsync(d_B, stream), "hipFreeAsync");
        cuda_check_error(hipFreeAsync(d_C, stream), "hipFreeAsync");
        cuda_check_error(hipEventRecord(e5, stream), "hipEventRecord");
        std::chrono::high_resolution_clock::time_point gpu_tp5{};
        cuda_check_error(hipStreamAddCallback(stream, report_completion_time_callback, &gpu_tp5, NULL_FLAGS), "hipStreamAddCallback");

        // Wait for stream to finish
        cuda_check_error(hipStreamSynchronize(stream), "hipStreamSynchronize");


        // Print execution time
        float gpu_step_dt1 = 0.0f, gpu_step_dt2 = 0.0f, gpu_step_dt3 = 0.0f, gpu_step_dt4 = 0.0f, gpu_step_dt5 = 0.0f;
        float gpu_total_dt1 = 0.0f, gpu_total_dt2 = 0.0f, gpu_total_dt3 = 0.0f, gpu_total_dt4 = 0.0f, gpu_total_dt5 = 0.0f;

        std::chrono::duration<double, std::milli> chrono_step_dt1 = gpu_tp1 - gpu_tp0;
        std::chrono::duration<double, std::milli> chrono_total_dt1 = gpu_tp1 - gpu_tp0;
        cuda_check_error(hipEventElapsedTime(&gpu_step_dt1, e0, e1), "hipEventElapsedTime");
        cuda_check_error(hipEventElapsedTime(&gpu_total_dt1, e0, e1), "hipEventElapsedTime");
        std::cout << " - CPU " << gpu_step_1 << ": " << chrono_step_dt1.count() << " ms (" << chrono_total_dt1.count() << " ms total)" << std::endl;
        std::cout << " - GPU " << gpu_step_1 << ": " << gpu_step_dt1 << " ms (" << gpu_total_dt1 << " ms total)" << std::endl;

        std::chrono::duration<double, std::milli> chrono_step_dt2 = gpu_tp2 - gpu_tp1;
        std::chrono::duration<double, std::milli> chrono_total_dt2 = gpu_tp2 - gpu_tp0;
        cuda_check_error(hipEventElapsedTime(&gpu_step_dt2, e1, e2), "hipEventElapsedTime");
        cuda_check_error(hipEventElapsedTime(&gpu_total_dt2, e0, e2), "hipEventElapsedTime");
        std::cout << " - CPU " << gpu_step_2 << ": " << chrono_step_dt2.count() << " ms (" << chrono_total_dt2.count() << " ms total)" << std::endl;
        std::cout << " - GPU " << gpu_step_2 << ": " << gpu_step_dt2 << " ms (" << gpu_total_dt2 << " ms total)" << std::endl;

        std::chrono::duration<double, std::milli> chrono_step_dt3 = gpu_tp3 - gpu_tp2;
        std::chrono::duration<double, std::milli> chrono_total_dt3 = gpu_tp3 - gpu_tp0;
        cuda_check_error(hipEventElapsedTime(&gpu_step_dt3, e2, e3), "hipEventElapsedTime");
        cuda_check_error(hipEventElapsedTime(&gpu_total_dt3, e0, e3), "hipEventElapsedTime");
        std::cout << " - CPU " << gpu_step_3 << ": " << chrono_step_dt3.count() << " ms (" << chrono_total_dt3.count() << " ms total)" << std::endl;
        std::cout << " - GPU " << gpu_step_3 << ": " << gpu_step_dt3 << " ms (" << gpu_total_dt3 << " ms total)" << std::endl;

        std::chrono::duration<double, std::milli> chrono_step_dt4 = gpu_tp4 - gpu_tp3;
        std::chrono::duration<double, std::milli> chrono_total_dt4 = gpu_tp4 - gpu_tp0;
        cuda_check_error(hipEventElapsedTime(&gpu_step_dt4, e3, e4), "hipEventElapsedTime");
        cuda_check_error(hipEventElapsedTime(&gpu_total_dt4, e0, e4), "hipEventElapsedTime");
        std::cout << " - CPU " << gpu_step_4 << ": " << chrono_step_dt4.count() << " ms (" << chrono_total_dt4.count() << " ms total)" << std::endl;
        std::cout << " - GPU " << gpu_step_4 << ": " << gpu_step_dt4 << " ms (" << gpu_total_dt4 << " ms total)" << std::endl;

        std::chrono::duration<double, std::milli> chrono_step_dt5 = gpu_tp5 - gpu_tp4;
        std::chrono::duration<double, std::milli> chrono_total_dt5 = gpu_tp5 - gpu_tp0;
        cuda_check_error(hipEventElapsedTime(&gpu_step_dt5, e4, e5), "hipEventElapsedTime");
        cuda_check_error(hipEventElapsedTime(&gpu_total_dt5, e0, e5), "hipEventElapsedTime");
        std::cout << " - CPU " << gpu_step_5 << ": " << chrono_step_dt5.count() << " ms (" << chrono_total_dt5.count() << " ms total)" << std::endl;
        std::cout << " - GPU " << gpu_step_5 << ": " << gpu_step_dt5 << " ms (" << gpu_total_dt5 << " ms total)" << std::endl;

        const auto cpu_tp0 = std::chrono::high_resolution_clock::now();

        std::cout << "CHECK WITH CPU:" << std::endl;
        const auto cpu_step_1 = "Convert data to Eigen";
        const Eigen::Map<Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>> A{h_A.data(), M, N};
        const Eigen::Map<Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>> B{h_B.data(), N, M};
        const Eigen::Map<Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>> C{h_C.data(), M, M};
        const auto cpu_tp1 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> cpu_step_dt1 = cpu_tp1 - cpu_tp0;
        std::chrono::duration<double, std::milli> cpu_total_dt1 = cpu_tp1 - cpu_tp0;
        std::cout << " - " << cpu_step_1 << ": " << cpu_step_dt1.count() << " ms (" << cpu_total_dt1.count() << " ms total)" << std::endl;

        const auto cpu_step_2 = "Compute result with Eigen";
        const auto C_cpu = A * B;
        const auto cpu_tp2 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> cpu_step_dt2 = cpu_tp2 - cpu_tp1;
        std::chrono::duration<double, std::milli> cpu_total_dt2 = cpu_tp2 - cpu_tp0;
        std::cout << " - " << cpu_step_2 << ": " << cpu_step_dt2.count() << " ms (" << cpu_total_dt2.count() << " ms total)" << std::endl;

        const auto cpu_step_3 = "Compute error matrix";
        const auto E = C - C_cpu;
        const auto cpu_tp3 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> cpu_step_dt3 = cpu_tp3 - cpu_tp2;
        std::chrono::duration<double, std::milli> cpu_total_dt3 = cpu_tp3 - cpu_tp0;
        std::cout << " - " << cpu_step_3 << ": " << cpu_step_dt3.count() << " ms (" << cpu_total_dt3.count() << " ms total)" << std::endl;

        const auto cpu_step_4 = "Compute error RMS";
        const auto E_rms = std::sqrt(E.squaredNorm() / size);
        const auto cpu_tp4 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> cpu_step_dt4 = cpu_tp4 - cpu_tp3;
        std::chrono::duration<double, std::milli> cpu_total_dt4 = cpu_tp4 - cpu_tp0;
        std::cout << " - " << cpu_step_4 << ": " << cpu_step_dt4.count() << " ms (" << cpu_total_dt4.count() << " ms total)" << std::endl;

        std::cout << "Error RMS: " << E_rms << std::endl;

        const auto tp_done = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> total_dt = tp_done - setup_tp0;
        std::cout << "DONE: " << total_dt.count() << " ms total" << std::endl;


    } catch (const cxxopts::exceptions::exception& e) {
        std::cerr << "Error parsing options: " << e.what() << std::endl;
        std::cout << options.help() << std::endl;
        return 1;
    }

    return 0;
}
