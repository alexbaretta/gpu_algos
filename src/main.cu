#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <random>
#include <chrono>

// Matrix dimensions
constexpr int M = 1000;    // Rows of first matrix
constexpr int N = 10000;   // Columns of first matrix / Rows of second matrix
constexpr int K = 1000;    // Columns of second matrix

// CUDA kernel for matrix multiplication
__global__ void matrixMultiply(const float* A, const float* B, float* C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            sum += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}

// Function to initialize matrix with random values
void initializeMatrix(std::vector<float>& matrix, int rows, int cols) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(0.0f, 1.0f);

    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] = dis(gen);
    }
}

int main() {
    // Allocate host memory
    std::vector<float> h_A(M * N);
    std::vector<float> h_B(N * K);
    std::vector<float> h_C(M * K);

    // Initialize matrices
    initializeMatrix(h_A, M, N);
    initializeMatrix(h_B, N, K);

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * N * sizeof(float));
    hipMalloc(&d_B, N * K * sizeof(float));
    hipMalloc(&d_C, M * K * sizeof(float));

    // Copy data to device
    hipMemcpy(d_A, h_A.data(), M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), N * K * sizeof(float), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((K + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    // Start timing
    auto start = std::chrono::high_resolution_clock::now();

    // Launch kernel
    matrixMultiply<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // End timing
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;

    // Copy result back to host
    hipMemcpy(h_C.data(), d_C, M * K * sizeof(float), hipMemcpyDeviceToHost);

    // Print execution time
    std::cout << "Matrix multiplication completed in " << duration.count() << " seconds" << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
